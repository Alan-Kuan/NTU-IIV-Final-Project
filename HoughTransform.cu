#include "hip/hip_runtime.h"
#include "HoughTransform.h"

#define STEP_SIZE 0.1
#define THRESHOLD 125
#define THETA_A 45.0
#define THETA_B 135.0
#define THETA_VARIATION 16.0

using namespace thrust;

class Line {
    
private:
    double theta;
    double rho;
    
public:
    
    __host__ __device__  Line(double theta, double rho){
        this->theta = theta;
        this->rho = rho;
    }
    
    __host__ __device__ double getY(double x) {
        double thetaRadian = (theta * PI) / 180.0;
        
        return (rho  - x * cos(thetaRadian)) / sin(thetaRadian);
    }
    
    __host__ __device__ double getX(double y) {
        double thetaRadian = (theta * PI) / 180.0;
        
        return (rho - y * sin(thetaRadian)) / cos(thetaRadian);
    }
};

void drawLine(Line line, Mat img);
extern void plotAccumulator(int nRows, int nCols, int *accumulator, const char *dest);
extern __host__ __device__ double calcRho(double x, double y, double theta);

void houghTransformSeq(VideoCapture capture, VideoWriter writer) {
	int nRows = (int) ceil(sqrt(FRAME_HEIGHT * FRAME_HEIGHT + FRAME_WIDTH * FRAME_WIDTH)) * 2;
	int nCols = 180 / STEP_SIZE;

	int *accumulator;
	accumulator = new int[nCols * nRows]();
	vector<Line> lines;

	Mat originalFrame, frame;

	clock_t loadTime = 0;
	clock_t prepTime = 0;
	clock_t houghTime = 0;
	clock_t drawTime = 0;
	clock_t t;

	for( ; ; ) {
		t = clock();
		capture >> originalFrame;
		loadTime += clock()-t;
		if(originalFrame.empty())
			break;

		t = clock();
		frame = applyPreprocessing(originalFrame);
		prepTime += clock()-t;

		t = clock();
		memset(accumulator, 0, nCols * nRows * sizeof(int));
		lines.clear();
        
        int rho;
        double theta;

		for(int i = 0; i < FRAME_HEIGHT; i++) {
			for (int j = 0; j < FRAME_WIDTH; j++) {
				if ((int) frame.at<uchar>(i, j) == 0)
					continue;
                
                // thetas of interest will be close to 45 and close to 135 (vertical lines)
                // we are doing 2 thetas at a time, 1 for each theta of Interest
                // we use thetas varying 15 degrees more and less
                for(int k = 0; k<2*THETA_VARIATION*(1/STEP_SIZE); k++){
                    theta = THETA_A-THETA_VARIATION + ((double)k*STEP_SIZE);
                    rho = calcRho(j, i, theta);
                    accumulator[(rho + (nRows / 2)) * nCols + (int)(theta/STEP_SIZE)] += 1;
                    
                    if (accumulator[(rho + (nRows / 2)) * nCols + (int)(theta/STEP_SIZE)] == THRESHOLD)
                        lines.push_back( Line(theta, rho));

                    theta = THETA_B-THETA_VARIATION + ((double)k*STEP_SIZE);
                    rho = calcRho(j, i, theta);
                    accumulator[(rho + (nRows / 2)) * nCols + (int)(theta/STEP_SIZE)] += 1;
                    
                    if (accumulator[(rho + (nRows / 2)) * nCols + (int)(theta/STEP_SIZE)] == THRESHOLD)
                        lines.push_back( Line(theta, rho));
                }
			}
		}
		houghTime += clock()-t;

		t = clock();
        for (int i = 0; i < lines.size(); i++)
            drawLine(lines[i], originalFrame);
		writer.write(originalFrame);
		drawTime += clock()-t;
	}
	cout<<"Read Time: "<<(((float)loadTime)/CLOCKS_PER_SEC)<<endl;
	cout<<"Prep Time: "<<(((float)prepTime)/CLOCKS_PER_SEC)<<endl;
	cout<<"Hough Time: "<<(((float)houghTime)/CLOCKS_PER_SEC)<<endl;
	cout<<"Write Time: "<<(((float)drawTime)/CLOCKS_PER_SEC)<<endl;
}

__global__ void houghKernel(unsigned char* frame, int nRows, int nCols, int *accumulator) {
	int i = blockIdx.x;
	int j = blockIdx.y;
	int theta = threadIdx.x * STEP_SIZE;
	int rho = calcRho(j, i, theta);

	if (((uchar) frame[(i * FRAME_WIDTH) + j]) != 0)
		atomicAdd(&accumulator[(rho + (nRows / 2)) * nCols + threadIdx.x], 1);
}

__global__ void houghKernel2(unsigned char* frame, int nRows, int nCols, int *accumulator) {
	int i = (blockIdx.x*blockDim.y)+threadIdx.y;
	int j = (blockIdx.y*blockDim.z)+threadIdx.z;
	double theta;
	int rho;

	if(i<FRAME_HEIGHT && j<FRAME_WIDTH && ((int) frame[(i * FRAME_WIDTH) + j]) != 0) {

		// thetas of interest will be close to 45 and close to 135 (vertical lines)
		// we are doing 2 thetas at a time, 1 for each theta of Interest
		// we use thetas varying 15 degrees more and less
		for(int k = threadIdx.x*(1/STEP_SIZE); k<(threadIdx.x+1)*(1/STEP_SIZE); k++){
			theta = THETA_A-THETA_VARIATION + ((double)k*STEP_SIZE);
			rho = calcRho(j, i, theta);
			atomicAdd(&accumulator[(rho + (nRows / 2)) * nCols + (int)(theta/STEP_SIZE)], 1);

			theta = THETA_B-THETA_VARIATION + ((double)k*STEP_SIZE);
			rho = calcRho(j, i, theta);
			atomicAdd(&accumulator[(rho + (nRows / 2)) * nCols + (int)(theta/STEP_SIZE)], 1);
		}
	}
}

__global__ void findLinesKernel(int nRows, int nCols, int *accumulator, Line *lines, int *lineCounter) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (accumulator[i * nCols + j] < THRESHOLD)
        return;

    for (int i_delta = -50; i_delta <= 50; i_delta++) {
        for (int j_delta = -50; j_delta <= 50; j_delta++) {
            if (i + i_delta > 0 && i + i_delta < nRows && j + j_delta > 0 && j + j_delta < nCols &&
                accumulator[(i + i_delta) * nCols + j + j_delta] > accumulator[i * nCols + j]) {
                return;
            }
        }
    }

    int insertPt = atomicAdd(lineCounter, 1);
    lines[insertPt%10] = Line(j*STEP_SIZE, i - (nRows / 2));
}

void houghTransformCuda(VideoCapture capture, VideoWriter writer) {
	int frameSize = FRAME_WIDTH * FRAME_HEIGHT * sizeof(uchar);
	int nRows = (int) ceil(sqrt(FRAME_HEIGHT * FRAME_HEIGHT + FRAME_WIDTH * FRAME_WIDTH)) * 2;
	int nCols = 180 / STEP_SIZE;

    Line *lines;
    lines = (Line *) malloc(10 * sizeof(Line));
    Line *d_lines;
    hipMalloc(&d_lines, 10 * sizeof(Line));
    int lineCounter = 0;
    int *d_lineCounter;
    hipMalloc(&d_lineCounter, sizeof(int));

	// device space for original image
	uchar *d_frame;
	hipMalloc<uchar>(&d_frame, frameSize);

	// device space for transformed image
	// TODO: we can reduce the the accumulator to accomodate only the thetas of interest
	int *d_accumulator;
	hipMalloc(&d_accumulator, nRows * nCols * sizeof(int));

	// kernell config 1
	// const dim3 block(180 / STEP_SIZE);
	// const dim3 grid(FRAME_HEIGHT, FRAME_WIDTH);
	// kernell config 2
	const dim3 block(32, 5, 5);
	const dim3 grid(ceil(FRAME_HEIGHT/5), ceil(FRAME_WIDTH/5));
    
    const dim3 findLinesBlock(32, 32);
    const dim3 findLinesGrid(ceil(nRows / 32), ceil(nCols / 32));

	Mat originalFrame, frame;

	clock_t loadTime = 0;
	clock_t prepTime = 0;
	clock_t houghTime = 0;
	clock_t drawTime = 0;
	clock_t t;

	for( ; ; ) {
		t = clock();
		capture >> originalFrame;
		loadTime += clock()-t;

		if(originalFrame.empty()){
			break;
		}

		t = clock();
		frame = applyPreprocessing(originalFrame);
		prepTime += clock()-t;

		t = clock();
		hipMemcpy(d_frame, frame.ptr(), frameSize, hipMemcpyHostToDevice);
		hipMemset(d_accumulator, 0, nRows * nCols * sizeof(int));

		houghKernel2<<<grid,block>>>(d_frame, nRows, nCols, d_accumulator);
		hipDeviceSynchronize();

		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString( err ));
        
        hipMemset(d_lineCounter, 0, sizeof(int));

        findLinesKernel<<<findLinesGrid, findLinesBlock>>>(nRows, nCols, d_accumulator, d_lines, d_lineCounter);
        hipDeviceSynchronize();
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error2: %s\n", hipGetErrorString( err ));
        
        hipMemcpy(&lineCounter, d_lineCounter, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(lines, d_lines, 10 * sizeof(Line), hipMemcpyDeviceToHost);

		houghTime += clock()-t;

		t = clock();
        for (int i = 0; i < lineCounter; i++)
            drawLine(lines[i], originalFrame);
        writer << originalFrame;
		drawTime += clock()-t;
	}

	cout<<"Read Time: "<<(((float)loadTime)/CLOCKS_PER_SEC)<<endl;
	cout<<"Prep Time: "<<(((float)prepTime)/CLOCKS_PER_SEC)<<endl;
	cout<<"Hough Time: "<<(((float)houghTime)/CLOCKS_PER_SEC)<<endl;
	cout<<"Write Time: "<<(((float)drawTime)/CLOCKS_PER_SEC)<<endl;

	hipFree(d_frame);
	hipFree(d_accumulator);
}

void drawLine(Line l, Mat img) {
    int y1 = img.rows;
    int y2 = (img.rows / 2) + (img.rows / 10);
    int x1 = (int) l.getX(y1);
    int x2 = (int) l.getX(y2);

    line(img, Point(x1, y1), Point(x2, y2), Scalar(255), 5, 8, 0);
}

/**
 * Plots 'accumulator' and saves created image to 'dest' (This is for debugging
 * purposes only
 */
void plotAccumulator(int nRows, int nCols, int *accumulator, const char *dest) {
	Mat plotImg(nRows, nCols, CV_8UC1, Scalar(0));
	for (int i = 0; i < nRows; i++) {
  		for (int j = 0; j < nCols; j++) {
			plotImg.at<uchar>(i, j) = min(accumulator[(i * nCols) + j], 255);
  		}
  	}

  	imwrite(dest, plotImg);
}

/**
 * Calculates rho based on the equation r = x cos(θ) + y sin(θ)
 *
 * @param x X coordinate of the pixel
 * @param y Y coordinate of the pixel
 * @param theta Angle between x axis and line connecting origin with closest
 * point on tested line
 *
 * @return Rho describing distance of origin to closest point on tested line
 */
__host__ __device__ double calcRho(double x, double y, double theta) {
	double thetaRadian = (theta * PI) / 180.0;

	return x * cos(thetaRadian) + y * sin(thetaRadian);
}

// __global__ void hough_kernel( unsigned char* img, int icols, int irows,
// 	int* hough, int nCols, int nRows)
// {
// //2D Index of current thread
// int theta = blockIdx.x;
// double thetaRad = ((double)theta*3.14159265358979323846)/180.0;

// for(int i = 0; i < icols; i++) {
// for(int j = 0; j < irows; j++) {
// //Location of gray pixel in output
// int img_id  = (j * icols) + i;

// if (((uchar) img[img_id]) == 0)
// continue;

// int rho = (((double)i*cos(thetaRad)) + ((double)j*sin(thetaRad)));

// hough[(rho + (nRows / 2)) * nCols + theta] += 1;

// }
// }

// }

// __global__ void hough_kernel2( unsigned char* img, int icols, int irows,
// 	int* hough, int nCols, int nRows)
// {
// //2D Index of current thread
// int theta = blockIdx.x;
// double thetaRad = ((double)theta*3.14159265358979323846)/180.0;
// double rho = blockIdx.y - (nRows/2);
// int j;

// for(int i = 0; i < icols; i++) {

// j = (int)((rho - (((double)i)*cos(thetaRad)))/sin(thetaRad));
// if(j>=irows || j<0)
// continue;

// if (((uchar) img[(j * icols) + i]) == 0)
// continue;


// hough[(blockIdx.y) * nCols + blockIdx.x] += 1;

// }
// }
