#include "hip/hip_runtime.h"
#include "HoughTransform.h"

#define STEP_SIZE 1
#define THRESHOLD 200

/**
 * Plots 'accumulator' and saves created image to 'dest' (This is for debugging
 * purposes only
 */
void plotAccumulator(int nRows, int nCols, int *accumulator, const char *dest) {
	Mat plotImg(nRows, nCols, CV_8UC1, Scalar(0));
	for (int i = 0; i < nRows; i++) {
  		for (int j = 0; j < nCols; j++) {
			plotImg.at<uchar>(i, j) = min(accumulator[(i * nCols) + j], 255);
  		}
  	}

  	imwrite(dest, plotImg);
}

/**
 * Calculates rho based on the equation r = x cos(θ) + y sin(θ)
 *
 * @param x X coordinate of the pixel
 * @param y Y coordinate of the pixel
 * @param theta Angle between x axis and line connecting origin with closest
 * point on tested line
 *
 * @return Rho describing distance of origin to closest point on tested line
 */
double calcRho(double x, double y, double theta) {
	double thetaRadian = (theta * PI) / 180.0;

	return x * cos(thetaRadian) + y * sin(thetaRadian);
}



/**
 * Performs sequential hough transform on given image
 *
 * @param img Input image on which hough transform is performed
 */
vector<Line> houghTransformSeq(Mat img) {
	int nRows = (int) ceil(sqrt(img.rows * img.rows + img.cols * img.cols)) * 2;
	int nCols = 180 / STEP_SIZE;

	int *accumulator;
	accumulator = new int[nCols * nRows]();
	vector<Line> lines;

	for(int i = 0; i < img.rows; i++) {
		for (int j = 0; j < img.cols; j++) {
   		if ((int) img.at<uchar>(i, j) == 0)
   			continue;

   		for (int k = 0; k < nCols; k++) {
   			double theta = ((double) k) * STEP_SIZE;

				int rho = calcRho(j, i, theta);

				accumulator[(rho + (nRows / 2)) * nCols + k] += 1;

				if(accumulator[(rho + (nRows / 2)) * nCols + k] == THRESHOLD)
					lines.push_back( Line(theta, rho));

   		}
		}
	}

	plotAccumulator(nRows, nCols, accumulator, "./res.jpg");

	return lines;

}

__global__ void hough_kernel( unsigned char* img, int width, int height, int grayWidthStep,
															int* hough, int nRows, int nCols)
{
	//2D Index of current thread
	double theta = ((double)threadIdx.x*3.14159265358979323846)/180.0;

	for(int i = 0; i < width; i++) {
		for(int j = 0; j < height; j++) {
			//Location of gray pixel in output
			int gray_tid  = j * grayWidthStep + i;

   		if (((int) img[gray_tid]) == 0)
   			continue;

			int rho = ((double)j*cos(theta) + (double)i*sin(theta));

			hough[(rho + (nRows / 2)) * nCols + threadIdx.x] += 1;

		}
	}

}

/**
 * Performs hough transform on given image using CUDA
 *
 * @param img Input image on which hough transform is performed
 */
vector<Line> houghTransformCuda(Mat img) {
	const int grayBytes = img.step * img.rows;
	vector<Line> lines;

	int nRows = (int) ceil(sqrt(img.rows * img.rows + img.cols * img.cols)) * 2;
	int nCols = 180 / STEP_SIZE;

	int *accumulator;
	accumulator = new int[nCols * nRows]();

	// device space for original image
	unsigned char *d_img;
	hipMalloc<unsigned char>(&d_img,grayBytes);
	hipMemcpy(d_img,img.ptr(),grayBytes,hipMemcpyHostToDevice);

	// device space for transformed image
	int *d_hough;
	hipMalloc(&d_hough, nRows*nCols*sizeof(int));
	hipMemcpy(d_hough,accumulator,nRows*nCols*sizeof(int),hipMemcpyHostToDevice);


	//Specify a reasonable block size
	const dim3 block(1, 1);

	//Calculate grid size to cover the whole image
	const dim3 grid(nCols, 1);

	hough_kernel<<<grid,block>>>(d_img, img.cols, img.rows, img.step, d_hough, nRows, nCols);

	hipDeviceSynchronize();

	hipMemcpy(accumulator,d_hough,nRows*nCols*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_img);
	hipFree(d_hough);

	for (int i = 0; i < nRows; i++) {
		for (int j = 0; j < nCols; j++) {
			if(accumulator[(i + (nRows / 2)) * nCols + j] == THRESHOLD)
				lines.push_back( Line(j, i));
		}
	}

	return lines;
}
